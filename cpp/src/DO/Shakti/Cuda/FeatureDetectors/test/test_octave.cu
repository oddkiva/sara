#include "hip/hip_runtime.h"
// ========================================================================== //
// This file is part of Sara, a basic set of libraries in C++ for computer
// vision.
//
// Copyright (C) 2020-present David Ok <david.ok8@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License v. 2.0. If a copy of the MPL was not distributed with this file,
// you can obtain one at http://mozilla.org/MPL/2.0/.
// ========================================================================== //

// Convolving in batch the input image does not seem very fast.
// Rather convolving sequentially seems much faster if we base ourselves from
// the computation time spent in the Halide implementation.


#define BOOST_TEST_MODULE "Shakti/CUDA/FeatureDetectors/Octave"

#include <boost/test/unit_test.hpp>

#include <DO/Sara/Core/Tensor.hpp>
#include <DO/Sara/ImageProcessing/LinearFiltering.hpp>

#include <DO/Shakti/Cuda/Utilities.hpp>
#include <DO/Shakti/Cuda/FeatureDetectors/Octave.hpp>


namespace sara = DO::Sara;
namespace shakti = DO::Shakti;
namespace sc = DO::Shakti::Cuda;


auto copy(sara::ImageView<float, 3>& src, sc::Octave<float>& dst)
{
  if (src.width() != dst.width() ||    //
      src.height() != dst.height() ||  //
      src.depth() != dst.scale_count())
    throw std::runtime_error{"Invalid sizes!"};

  auto copy_params = hipMemcpy3DParms{};
  {
    copy_params.srcPtr = make_hipPitchedPtr(                   //
        reinterpret_cast<void*>(src.data()),                    //
        src.width() * sizeof(float), src.width(), src.height()  //
    );
    copy_params.srcPos = make_hipPos(0, 0, 0);

    copy_params.dstArray = dst;
    copy_params.dstPos = make_hipPos(0, 0, 0);

    // Because we use a CUDA array the extent is in terms of number of elements
    // and not in bytes.
    copy_params.extent =
        make_hipExtent(src.width(), src.height(), src.depth());
    copy_params.kind = hipMemcpyHostToDevice;
  }

  SHAKTI_SAFE_CUDA_CALL(hipMemcpy3D(&copy_params));
}

auto copy(sc::Octave<float>& src, sara::ImageView<float, 3>& dst)
{
  if (src.width() != dst.width() ||    //
      src.height() != dst.height() ||  //
      src.scale_count() != dst.depth())
    throw std::runtime_error{"Invalid sizes!"};

  auto copy_params = hipMemcpy3DParms{};
  {
    copy_params.srcArray = src;
    copy_params.srcPos = make_hipPos(0, 0, 0);
    copy_params.dstPtr = make_hipPitchedPtr(                   //
        reinterpret_cast<void*>(dst.data()),                    //
        dst.width() * sizeof(float), dst.width(), dst.height()  //
    );
    copy_params.dstPos = make_hipPos(0, 0, 0);

    // Because we use a CUDA array the extent is in terms of number of elements
    // and not in bytes.
    copy_params.extent =
        make_hipExtent(src.width(), src.height(), src.scale_count());
    copy_params.kind = hipMemcpyDeviceToHost;
  }

  SHAKTI_SAFE_CUDA_CALL(hipMemcpy3D(&copy_params));
}

BOOST_AUTO_TEST_CASE(test_octave_with_different_data_types)
{
  static constexpr auto w = 1920;
  static constexpr auto h = 1080;
  static constexpr auto scale_count = 3;

  const auto octave_16u =
      sc::make_gaussian_octave<std::uint16_t>(w, h, scale_count);
  const auto octave_16f = sc::make_gaussian_octave<half>(w, h, scale_count);
  const auto octave_32f = sc::make_gaussian_octave<half>(w, h, scale_count);

  // DOES NOT WORK.
  // auto octave_64f = DO::Shakti::Cuda::make_gaussian_octave<double>(w, h);
  BOOST_CHECK_EQUAL(octave_16u.scale_count(), 6);
  BOOST_CHECK_EQUAL(octave_16f.scale_count(), 6);
}

BOOST_AUTO_TEST_CASE(test_copy)
{
  static constexpr auto w = 11;
  static constexpr auto h = 11;
  static constexpr auto scale_count = 1;

  // Initialize the octave CUDA surface.
  auto octave_32f = sc::make_gaussian_octave<float>(w, h, scale_count);
  octave_32f.init_surface();
  BOOST_CHECK_EQUAL(octave_32f.scale_count(), scale_count + 3);

  auto values = sara::Image<float, 3>{w, h, octave_32f.scale_count()};
  values.flat_array().fill(1);

  auto values2 = sara::Image<float, 3>{w, h, octave_32f.scale_count()};
  values2.flat_array().fill(0);

  copy(values, octave_32f);
  copy(octave_32f, values2);
  BOOST_CHECK(values == values2);
}


__global__ void fill(hipSurfaceObject_t output, int width, int height,
                     int scale_count)
{
  // Calculate normalized texture coordinates
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < width && y < height && z < scale_count)
  {
    const float out = z * width * height + y * width + x;
    surf2DLayeredwrite(out, output, x * sizeof(float), y, z);
  }
}

BOOST_AUTO_TEST_CASE(test_fill)
{
  static constexpr auto w = 3;
  static constexpr auto h = 5;
  static constexpr auto scale_count = 1;

  // Initialize the octave CUDA surface.
  auto octave = sc::make_gaussian_octave<float>(w, h, scale_count);
  octave.init_surface();

  // Initialize the octave in CUDA.
  const dim3 threadsperBlock(16, 16, 2);
  const dim3 numBlocks(
      (octave.width() + threadsperBlock.x - 1) / threadsperBlock.x,
      (octave.height() + threadsperBlock.y - 1) / threadsperBlock.y,
      (octave.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
  fill<<<numBlocks, threadsperBlock>>>(octave.surface_object(),  //
                                       octave.width(), octave.height(),
                                       octave.scale_count()  //
  );

  // Check the result.
  auto result = sara::Image<float, 3>{w, h, octave.scale_count()};
  result.flat_array().fill(0);
  copy(octave, result);

  auto gt = sara::Image<float, 3>{w, h, octave.scale_count()};
  for (auto i = 0u; i < gt.size(); ++i)
    gt.data()[i] = i;

  // for (auto s = 0; s < result.depth(); ++s)
  //   SARA_DEBUG << s << "\n"
  //              << sara::tensor_view(result)[s].matrix() << std::endl;
  BOOST_CHECK(result == gt);
}


static constexpr auto max_thread_count = 1024;
static constexpr auto tile_x = 32;
static constexpr auto tile_y = 32;
static constexpr auto tile_z = max_thread_count / tile_x / tile_y;

__constant__ float constant_gauss_kernels[512];
__constant__ int constant_gauss_kernel_sizes[16];
__constant__ int constant_gauss_kernel_radius;
__constant__ int constant_kernel_count;
__constant__ int constant_kernel_size;


__global__ void convolve_x(hipSurfaceObject_t input,   //
                           hipSurfaceObject_t output,  //
                           int input_layer,             //
                           int width, int height,       //
                           int scale_count)
{
  // Calculate normalized texture coordinates
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < width && y < height && z < scale_count)
  {
    auto out = float{};

#pragma unroll
    for (auto k = 0; k <= constant_kernel_size; ++k)
    {
      float val;
      surf2DLayeredread(                                           //
          &val,                                                    //
          input,                                                   //
          (x - constant_gauss_kernel_radius + k) * sizeof(float),  //
          y,                                                       //
          input_layer,                                             //
          hipBoundaryModeClamp);
      out += constant_gauss_kernels[z * constant_kernel_size + k] * val;
    }

    surf2DLayeredwrite<float>(out, output, x * sizeof(float), y, z);
  }
}

__global__ void convolve_y(hipSurfaceObject_t input,   //
                           hipSurfaceObject_t output,  //
                           int width, int height,       //
                           int scale_count)
{
  // Calculate normalized texture coordinates
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < width && y < height && z < scale_count)
  {
    auto out = float{};

#pragma unroll
    for (auto k = 0; k <= constant_kernel_size; ++k)
    {
      float val;
      surf2DLayeredread(                         //
          &val,                                  //
          input,                                 //
          x * sizeof(float),                     //
          y - constant_gauss_kernel_radius + k,  //
          z,                                     //
          hipBoundaryModeClamp);
      out += constant_gauss_kernels[z * constant_kernel_size + k] * val;
    }

    surf2DLayeredwrite<float>(out, output, x * sizeof(float), y, z);
  }
}

BOOST_AUTO_TEST_CASE(test_convolve)
{
  std::cout << shakti::get_devices().front() << std::endl;

  static constexpr auto scale_count = 3;
  static constexpr auto scale_camera = 1.f;
  static constexpr auto scale_initial = 1.6f;
  static constexpr auto gaussian_truncation_factor = 4.f;
  static const float scale_factor = std::pow(2.f, 1.f / scale_count);

  // Set up the list of scales in the discrete octave.
  auto scales = std::vector<float>(scale_count + 3);
  for (auto i = 0; i < scale_count + 3; ++i)
    scales[i] = scale_initial * std::pow(scale_factor, i);

  // Calculate the Gaussian smoothing values.
  auto sigmas = std::vector<float>(scale_count + 3);
  for (auto i = 0u; i < sigmas.size(); ++i)
    sigmas[i] = std::sqrt(std::pow(scales[i], 2) - std::pow(scale_camera, 2));

  auto sigmaDeltas = std::vector<float>(scale_count + 3);
  for (auto i = 0u; i < sigmas.size(); ++i)
    sigmaDeltas[i] =
        i == 0 ? std::sqrt(std::pow(scales[0], 2) - std::pow(scale_camera, 2))
               : std::sqrt(std::pow(scales[i], 2) - std::pow(scales[i - 1], 2));
  SARA_CHECK(
      Eigen::Map<const Eigen::RowVectorXf>(sigmas.data(), sigmas.size()));
  SARA_CHECK(Eigen::Map<const Eigen::RowVectorXf>(sigmaDeltas.data(),
                                                  sigmaDeltas.size()));

  // Calculater the kernel dimensions.
  auto kernel_sizes = std::vector<int>{};
  for (const auto& sigma : sigmas)
  {
    const auto radius = static_cast<int>(               //
        std::round(gaussian_truncation_factor * sigma)  //
    );
    kernel_sizes.push_back(2 * radius + 1);
  }

  const auto kernel_size_max = kernel_sizes.back();
  const auto kernel_radius = kernel_size_max / 2;

  // Fill the Gaussian kernels.
  auto kernels = sara::Tensor_<float, 2>{
      scale_count + 3,  //
      kernel_size_max   //
  };
  kernels.flat_array().fill(0);

  for (auto n = 0; n < kernels.size(0); ++n)
  {
    const auto& sigma = sigmas[n];
    const auto ksize = kernel_sizes[n];
    const auto kradius = ksize / 2;
    const auto two_sigma_squared = 2 * sigma * sigma;

    for (auto k = 0; k < ksize; ++k)
      kernels(n, k + kernel_radius - kradius) =
          exp(-std::pow(k - kradius, 2) / two_sigma_squared);

    const auto kernel_sum =
        std::accumulate(&kernels(n, kernel_radius - kradius),
                        &kernels(n, kernel_radius - kradius) + ksize, 0.f);

    for (auto k = 0; k < ksize; ++k)
      kernels(n, k + kernel_radius - kradius) /= kernel_sum;
  }

  Eigen::IOFormat HeavyFmt(3, 0, ", ", ",\n", "[", "]", "[", "]");
  SARA_CHECK(Eigen::Map<const Eigen::RowVectorXf>(  //
      sigmas.data(),                                //
      sigmas.size())                                //
  );
  SARA_CHECK(kernels.sizes().reverse().transpose());
  SARA_DEBUG << "stacked kernels =\n"
             << kernels.matrix().transpose().format(HeavyFmt) << std::endl;


  SARA_DEBUG << "Copying the stacked kernels to CUDA constant memory"
             << std::endl;
  shakti::tic();
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constant_gauss_kernels),  //
                                           kernels.data(),
                                           kernels.size() * sizeof(float)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constant_gauss_kernel_sizes),
                                           kernel_sizes.data(),
                                           kernel_sizes.size() * sizeof(int)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constant_kernel_count),  //
                                           kernels.sizes().data(),
                                           sizeof(int)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constant_kernel_size),  //
                                           kernels.sizes().data() + 1,
                                           sizeof(int)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constant_gauss_kernel_radius),  //
                                           &kernel_radius, sizeof(int)));
  shakti::toc("copy to constant memory");

#define THIS_WORKS
#ifdef THIS_WORKS
  auto kernels_copied = sara::Tensor_<float, 2>{kernels.sizes()};
  kernels_copied.flat_array().fill(-1);
  SARA_DEBUG << "kernels copied (initialized)=\n"
             << kernels_copied.matrix().transpose().format(HeavyFmt)
             << std::endl;

  SHAKTI_SAFE_CUDA_CALL(hipMemcpyFromSymbol(kernels_copied.data(),
                                             constant_gauss_kernels,
                                             kernels.size() * sizeof(float)));
  SARA_DEBUG << "kernels copied=\n"
             << kernels_copied.matrix().transpose().format(HeavyFmt)
             << std::endl;

  auto kernel_size = int{};
  auto kernel_count = int{};
  auto kernel_radius_point_copied = int{};
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyFromSymbol(&kernel_size,  HIP_SYMBOL(//
                                             constant_kernel_size),
                                             sizeof(int)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyFromSymbol(&kernel_count,  HIP_SYMBOL(//
                                             constant_kernel_count),
                                             sizeof(int)));
  SHAKTI_SAFE_CUDA_CALL(hipMemcpyFromSymbol(&kernel_radius_point_copied,  HIP_SYMBOL(//
                                             constant_gauss_kernel_radius),
                                             sizeof(int)));

  SARA_CHECK(kernel_size);
  SARA_CHECK(kernel_count);
  SARA_CHECK(kernel_radius_point_copied);
#endif

  const auto w = 4 * 1920;
  const auto h = 4 * 1080;

  // Initialize the octave CUDA surface.
  auto octave = sc::make_gaussian_octave<float>(w, h, scale_count);
  octave.init_surface();
  BOOST_CHECK_EQUAL(octave.scale_count(), scale_count + 3);

  // Initialize the octave.
  auto dirac = sara::Image<float, 3>{w, h, octave.scale_count()};
  dirac.flat_array().fill(0);
  for (auto s = 0; s < dirac.depth(); ++s)
    dirac(w / 2, h / 2, s) = 1;
  copy(dirac, octave);

  // Convolve the octave.
  auto conv_x = sc::make_gaussian_octave<float>(w, h, scale_count);
  auto conv_y = sc::make_gaussian_octave<float>(w, h, scale_count);
  conv_x.init_surface();
  conv_y.init_surface();

  shakti::tic();
  {
    const auto threadsperBlock = dim3(tile_x, tile_y, tile_z);
    const auto numBlocks = dim3(
        (octave.width() + threadsperBlock.x - 1) / threadsperBlock.x,
        (octave.height() + threadsperBlock.y - 1) / threadsperBlock.y,
        (octave.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);

    // x-convolution.
    convolve_x<<<numBlocks, threadsperBlock>>>(                //
        octave.surface_object(),                               //
        conv_x.surface_object(),                               //
        /* input_layer */ 0,                                   //
        octave.width(), octave.height(), octave.scale_count()  //
    );
  }

  // y-convolution.
  {
    const auto threadsperBlock = dim3(tile_x, tile_y, tile_z);
    const auto numBlocks = dim3(
        (octave.width() + threadsperBlock.x - 1) / threadsperBlock.x,
        (octave.height() + threadsperBlock.y - 1) / threadsperBlock.y,
        (octave.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);

    convolve_y<<<numBlocks, threadsperBlock>>>(                //
        conv_x.surface_object(),                               //
        conv_y.surface_object(),                               //
        octave.width(), octave.height(), octave.scale_count()  //
    );
  }
  shakti::toc("Gaussian convolution");


  auto values = dirac;
  copy(conv_y, values);

  if (w < 10 && h < 10)
  {
    for (auto s = 0; s < values.depth(); ++s)
    {
      SARA_CHECK(s);
      std::cout << sara::tensor_view(values)[s].matrix().format(HeavyFmt)
                << std::endl;
    }
  }
}
