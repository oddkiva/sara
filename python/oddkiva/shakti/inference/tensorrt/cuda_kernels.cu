
#include <hip/hip_runtime.h>
using uint8_t = unsigned char;


__global__ auto from_hwc_uint8_to_chw_float(float* out, const uint8_t* in,
                                            const int w, const int h) -> void
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= w || y >= h)
    return;

  const auto wh = w * h;
  const int gi_out = y * w + x;
  const int gi_in = 3 * (y * w + x);
  // clang-format off
  out[/* 0 *  wh */ + gi_out] = float(in[gi_in + 0]);
  out[/* 1 */ wh    + gi_out] = float(in[gi_in + 1]);
  out[   2 *  wh    + gi_out] = float(in[gi_in + 2]);
  // clang-format on
}

__global__ auto naive_downsample(float* out, const float* in, const int wout,
                                 const int hout, const int win, const int hin)
    -> void
{
  const int xout = blockIdx.x * blockDim.x + threadIdx.x;
  const int yout = blockIdx.y * blockDim.y + threadIdx.y;
  const int c = blockIdx.z * blockDim.z + threadIdx.z;

  if (xout >= wout || yout >= hout || c >= 3)
    return;

  const float sx = float(win) / float(wout);
  const float sy = float(hin) / float(hout);

  int xin = int(xout * sx + 0.5f);
  int yin = int(yout * sy + 0.5f);

  if (xin >= win)
    xin = win - 1;
  if (yin >= hin)
    yin = hin - 1;

  const int gi_out = c * wout * hout + yout * wout + xout;
  const int gi_in = c * win * hin + yin * win + xin;
  out[gi_out] = in[gi_in];
}


