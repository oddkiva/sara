#include "hip/hip_runtime.h"
// ========================================================================== //
// This file is part of Sara, a basic set of libraries in C++ for computer
// vision.
//
// Copyright (C) 2020-present David Ok <david.ok8@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License v. 2.0. If a copy of the MPL was not distributed with this file,
// you can obtain one at http://mozilla.org/MPL/2.0/.
// ========================================================================== //

// Convolving in batch the input image does not seem very fast.
// Rather convolving sequentially seems much faster if we base ourselves from
// the computation time spent in the Halide implementation.


#define BOOST_TEST_MODULE "Shakti/CUDA/FeatureDetectors/Octave"

#include <boost/test/unit_test.hpp>

#include <DO/Sara/Core/Tensor.hpp>
#include <DO/Sara/ImageProcessing/LinearFiltering.hpp>

#include <DO/Shakti/Cuda/FeatureDetectors/Octave.hpp>
#include <DO/Shakti/Cuda/Utilities.hpp>


namespace sara = DO::Sara;
namespace shakti = DO::Shakti;
namespace sc = DO::Shakti::Cuda;


__global__ void fill(hipSurfaceObject_t output, int width, int height,
                     int scale_count)
{
  // Calculate normalized texture coordinates
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < width && y < height && z < scale_count)
  {
    const float out = z * width * height + y * width + x;
    surf2DLayeredwrite(out, output, x * sizeof(float), y, z);
  }
}

BOOST_AUTO_TEST_CASE(test_fill)
{
  static constexpr auto w = 3;
  static constexpr auto h = 5;
  static constexpr auto scale_count = 1;

  // Initialize the octave CUDA surface.
  auto octave = sc::make_gaussian_octave<float>(w, h, scale_count);
  octave.init_surface();

  // Initialize the octave in CUDA.
  const dim3 threadsperBlock(16, 16, 2);
  const dim3 numBlocks(
      (octave.width() + threadsperBlock.x - 1) / threadsperBlock.x,
      (octave.height() + threadsperBlock.y - 1) / threadsperBlock.y,
      (octave.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
  fill<<<numBlocks, threadsperBlock>>>(octave.surface_object().value(),  //
                                       octave.width(), octave.height(),
                                       octave.scale_count());

  // Check the result.
  auto result = sara::Image<float, 3>{w, h, octave.scale_count()};
  result.flat_array().fill(0);
  octave.array().copy_to(result);

  auto gt = sara::Image<float, 3>{w, h, octave.scale_count()};
  for (auto i = 0u; i < gt.size(); ++i)
    gt.data()[i] = i;

  // for (auto s = 0; s < result.depth(); ++s)
  //   SARA_DEBUG << s << "\n"
  //              << sara::tensor_view(result)[s].matrix() << std::endl;
  BOOST_CHECK(result == gt);
}
