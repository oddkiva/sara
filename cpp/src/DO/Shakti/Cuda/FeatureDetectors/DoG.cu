#include "hip/hip_runtime.h"
// ========================================================================== //
// This file is part of Shakti, a basic set of CUDA accelerated libraries in
// C++ for computer vision.
//
// Copyright (C) 2022-present David Ok <david.ok8@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License v. 2.0. If a copy of the MPL was not distributed with this file,
// you can obtain one at http://mozilla.org/MPL/2.0/.
// ========================================================================== //

#include <DO/Shakti/Cuda/FeatureDetectors/DoG.hpp>


namespace DO::Shakti::Cuda {

  __global__ auto dog(hipSurfaceObject_t gaussian_octave,
                      hipSurfaceObject_t dog_octave,  //
                      int w, int h, int d) -> void
  {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= w || y >= h || z >= d)
      return;

    float val1;
    surf2DLayeredread(&val1, gaussian_octave, x * sizeof(float), y, z,
                      hipBoundaryModeClamp);

    float val2;
    surf2DLayeredread(&val2, gaussian_octave, x * sizeof(float), y, z + 1,
                      hipBoundaryModeClamp);

#ifdef USE_SMEM
    __shared__ float sdata[32][32][2];

    // No real benefit in using the shared memory with a 4K video:
    //
    // [DoG] Elapsed time = 1.44467 ms
    const auto& tx = threadIdx.x;
    const auto& ty = threadIdx.y;
    sdata[ty][tx][0] = val1;
    sdata[ty][tx][1] = val2;
    __syncthreads();

    const float diff = sdata[ty][tx][1] - sdata[ty][tx][0];
#else
    // [DoG] Elapsed time = 1.42746 ms
    const float diff = val2 - val1;
#endif
    surf2DLayeredwrite(diff, dog_octave, x * sizeof(float), y, z);
  }

  // TODO: speed this up as it is very slow.
  __global__ auto local_scale_space_extremum(hipSurfaceObject_t dog_octave,
                                             std::int8_t* ext_map,  //
                                             int dog_w, int dog_h, int dog_d,
                                             int ext_pitch) -> void
  {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= dog_w || y >= dog_h || z >= dog_d)
      return;

    const auto gi = (z * dog_h + y) * ext_pitch + x;

    if (x == 0 || y == 0 || z == 0 ||  //
        x == dog_w - 1 || y == dog_h - 1 || z == dog_d - 1)
    {
      ext_map[gi] = 0;
      return;
    }

    float val;
    surf2DLayeredread(&val, dog_octave, x * sizeof(float), y, z,
                      hipBoundaryModeClamp);

    float val_ext;
    if (val > 0)
    {
#pragma unroll
      for (auto dz = -1; dz <= 1; ++dz)
      {
#pragma unroll
        for (auto dy = -1; dy <= 1; ++dy)
        {
#pragma unroll
          for (auto dx = -1; dx <= 1; ++dx)
          {
            // if (dz == 0 && dy == 0 && dx == 0)
            //   continue;

            float val1;
            surf2DLayeredread(&val1, dog_octave, (x + dx) * sizeof(float),
                              y + dy, z + dz, hipBoundaryModeClamp);

            val_ext = max(val1, val_ext);
          }
        }
      }
    }
    else
    {
#pragma unroll
      for (auto dz = -1; dz <= 1; ++dz)
      {
#pragma unroll
        for (auto dy = -1; dy <= 1; ++dy)
        {
#pragma unroll
          for (auto dx = -1; dx <= 1; ++dx)
          {
            float val1;
            surf2DLayeredread(&val1, dog_octave, (x + dx) * sizeof(float),
                              y + dy, z + dz, hipBoundaryModeClamp);

            val_ext = min(val1, val_ext);
          }
        }
      }
    }

    auto extremum_type = std::int8_t{};
    if (val == val_ext && val > 0)
      extremum_type = 1;
    else if (val == val_ext && val < 0)
      extremum_type = -1;

    ext_map[gi] = extremum_type;
  }

  auto compute_dog_octave(const Octave<float>& gaussians, Octave<float>& dogs)
      -> void
  {
    if (dogs.width() != gaussians.width() ||
        dogs.height() != gaussians.height() ||
        dogs.scale_count() != gaussians.scale_count() - 1)
      throw std::runtime_error{"Invalid octave sizes!"};

    if (!gaussians.surface_object().initialized())
      throw std::runtime_error{"Gaussian surface object is uninitialized!"};

    if (!dogs.surface_object().initialized())
      dogs.init_surface();

    static constexpr auto threadsperBlock = dim3(32, 32, 1);
    static const auto numBlocks =
        dim3((dogs.width() + threadsperBlock.x - 1) / threadsperBlock.x,
             (dogs.height() + threadsperBlock.y - 1) / threadsperBlock.y,
             (dogs.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
    dog<<<numBlocks, threadsperBlock>>>(gaussians.surface_object(),
                                        dogs.surface_object(), dogs.width(),
                                        dogs.height(), dogs.scale_count());
  }

  auto compute_scale_space_extremum_map(
      const Octave<float>& dogs,
      MultiArrayView<std::int8_t, 3, RowMajorStrides>& extremum_map) -> void
  {
    if (extremum_map.width() != dogs.width() ||
        extremum_map.height() != dogs.height() ||
        extremum_map.depth() != dogs.scale_count())
      throw std::runtime_error{"Invalid octave sizes!"};

    if (!dogs.surface_object().initialized())
      throw std::runtime_error{"DoG surface object is uninitialized!"};

    static constexpr auto threadsperBlock = dim3(32, 16, 2);
    static const auto numBlocks =
        dim3((dogs.width() + threadsperBlock.x - 1) / threadsperBlock.x,
             (dogs.height() + threadsperBlock.y - 1) / threadsperBlock.y,
             (dogs.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
    local_scale_space_extremum<<<numBlocks, threadsperBlock>>>(
        dogs.surface_object(), extremum_map.data(),       //
        dogs.width(), dogs.height(), dogs.scale_count(),  //
        extremum_map.padded_width());
  }

}  // namespace DO::Shakti::Cuda
