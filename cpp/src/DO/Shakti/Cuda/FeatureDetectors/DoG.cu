#include "hip/hip_runtime.h"
// ========================================================================== //
// This file is part of Shakti, a basic set of CUDA accelerated libraries in
// C++ for computer vision.
//
// Copyright (C) 2022-present David Ok <david.ok8@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License v. 2.0. If a copy of the MPL was not distributed with this file,
// you can obtain one at http://mozilla.org/MPL/2.0/.
// ========================================================================== //

#include <DO/Shakti/Cuda/FeatureDetectors/DoG.hpp>


namespace DO::Shakti::Cuda {

  __global__ auto dog(hipSurfaceObject_t gaussian_octave,
                      hipSurfaceObject_t dog_octave,  //
                      int w, int h, int d) -> void
  {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= w || y >= h || z >= d)
      return;

    float val1;
    surf2DLayeredread(&val1, gaussian_octave, x * sizeof(float), y, z,
                      hipBoundaryModeClamp);

    float val2;
    surf2DLayeredread(&val2, gaussian_octave, x * sizeof(float), y, z + 1,
                      hipBoundaryModeClamp);

#ifdef USE_SHARED_MEMORY
    __shared__ float sdata[32][32][2];

    // No real benefit in using the shared memory with a 4K video:
    //
    // [DoG] Elapsed time = 1.44467 ms
    const auto& tx = threadIdx.x;
    const auto& ty = threadIdx.y;
    sdata[ty][tx][0] = val1;
    sdata[ty][tx][1] = val2;
    __syncthreads();

    const float diff = sdata[ty][tx][1] - sdata[ty][tx][0];
#else
    // [DoG] Elapsed time = 1.42746 ms
    const float diff = val2 - val1;
#endif
    surf2DLayeredwrite(diff, dog_octave, x * sizeof(float), y, z);
  }

  auto compute_dog_octave(const Octave<float>& gaussians, Octave<float>& dogs)
      -> void
  {
    if (dogs.width() != gaussians.width() ||
        dogs.height() != gaussians.height() ||
        dogs.scale_count() != gaussians.scale_count() - 1)
      throw std::runtime_error{"Invalid octave sizes!"};

    if (!gaussians.surface_object().initialized())
      throw std::runtime_error{"Gaussian surface object is uninitialized!"};

    if (!dogs.surface_object().initialized())
      dogs.init_surface();

    static constexpr auto threadsperBlock = dim3(32, 32, 1);
    static const auto numBlocks =
        dim3((dogs.width() + threadsperBlock.x - 1) / threadsperBlock.x,
             (dogs.height() + threadsperBlock.y - 1) / threadsperBlock.y,
             (dogs.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
    dog<<<numBlocks, threadsperBlock>>>(gaussians.surface_object(),
                                        dogs.surface_object(), dogs.width(),
                                        dogs.height(), dogs.scale_count());
  }

}  // namespace DO::Shakti::Cuda
