#include "hip/hip_runtime.h"
// ========================================================================== //
// This file is part of Shakti, a basic set of CUDA accelerated libraries in
// C++ for computer vision.
//
// Copyright (C) 2022-present David Ok <david.ok8@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License v. 2.0. If a copy of the MPL was not distributed with this file,
// you can obtain one at http://mozilla.org/MPL/2.0/.
// ========================================================================== //

#include <DO/Shakti/Cuda/FeatureDetectors/DoG.hpp>


namespace DO::Shakti::Cuda {

  __global__ auto dog(hipSurfaceObject_t gaussian_octave,
                      hipSurfaceObject_t dog_octave,  //
                      int w, int h, int d) -> void
  {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= w || y >= h || z >= d)
      return;

    float val1;
    surf2DLayeredread(&val1, gaussian_octave, x * sizeof(float), y, z,
                      hipBoundaryModeClamp);

    float val2;
    surf2DLayeredread(&val2, gaussian_octave, x * sizeof(float), y, z + 1,
                      hipBoundaryModeClamp);

#ifdef USE_SHARED_MEMORY
    __shared__ float sdata[32][32][2];

    // No real benefit in using the shared memory with a 4K video:
    //
    // [DoG] Elapsed time = 1.44467 ms
    const auto& tx = threadIdx.x;
    const auto& ty = threadIdx.y;
    sdata[ty][tx][0] = val1;
    sdata[ty][tx][1] = val2;
    __syncthreads();

    const float diff = sdata[ty][tx][1] - sdata[ty][tx][0];
#else
    // [DoG] Elapsed time = 1.42746 ms
    const float diff = val2 - val1;
#endif
    surf2DLayeredwrite(diff, dog_octave, x * sizeof(float), y, z);
  }

  // TODO: speed this up as it is very slow.
  static constexpr auto tile_x = 32;
  static constexpr auto tile_y = 16;
  static constexpr auto tile_z = 2;
  __global__ auto local_scale_space_extremum(hipSurfaceObject_t dog_octave,
                                             std::int8_t* ext_map,  //
                                             int dog_w, int dog_h, int dog_d,
                                             int ext_pitch,
                                             float min_extremum_abs_value,
                                             float edge_ratio_thres) -> void
  {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= dog_w || y >= dog_h || z >= dog_d)
      return;

    const auto gi = (z * dog_h + y) * ext_pitch + x;

    if (x == 0 || y == 0 || z == 0 ||  //
        x == dog_w - 1 || y == dog_h - 1 || z == dog_d - 1)
    {
      ext_map[gi] = 0;
      return;
    }

    // Avoid the local extremum loops.
    float val;
    surf2DLayeredread(&val, dog_octave, x * sizeof(float), y, z,
                      hipBoundaryModeClamp);
    if (abs(val) < 0.8f * min_extremum_abs_value)  // 0.8f prefiltering ratio.
    {
      ext_map[gi] = 0;
      return;
    }

    // Use the shared memory to fully leverage the GPU speed.
    __shared__ float s_prev[tile_z][tile_y + 2][tile_x + 2];
    __shared__ float s_curr[tile_z][tile_y + 2][tile_x + 2];
    __shared__ float s_next[tile_z][tile_y + 2][tile_x + 2];

    const auto& tx = threadIdx.x;
    const auto& ty = threadIdx.y;
    const auto& tz = threadIdx.z;

    // Populate the previous scale.
    //
    // Top-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y - 1, z - 1,
                      hipBoundaryModeClamp);
    s_prev[tz][ty + 0][tx + 0] = val;
    // Top-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y - 1, z - 1,
                      hipBoundaryModeClamp);
    s_prev[tz][ty + 0][tx + 2] = val;
    // Bottom-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y + 1, z - 1,
                      hipBoundaryModeClamp);
    s_prev[tz][ty + 2][tx + 0] = val;
    // Bottom-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y + 1, z - 1,
                      hipBoundaryModeClamp);
    s_prev[tz][ty + 2][tx + 2] = val;

    // Populate the current scale.
    //
    // Top-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y - 1, z,
                      hipBoundaryModeClamp);
    s_curr[tz][ty + 0][tx + 0] = val;
    // Top-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y - 1, z,
                      hipBoundaryModeClamp);
    s_curr[tz][ty + 0][tx + 2] = val;
    // Bottom-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y + 1, z,
                      hipBoundaryModeClamp);
    s_curr[tz][ty + 2][tx + 0] = val;
    // Bottom-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y + 1, z,
                      hipBoundaryModeClamp);
    s_curr[tz][ty + 2][tx + 2] = val;

    // Populate the next scale.
    //
    // Top-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y - 1, z + 1,
                      hipBoundaryModeClamp);
    s_next[tz][ty + 0][tx + 0] = val;
    // Top-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y - 1, z + 1,
                      hipBoundaryModeClamp);
    s_next[tz][ty + 0][tx + 2] = val;
    // Bottom-left
    surf2DLayeredread(&val, dog_octave, (x - 1) * sizeof(float), y + 1, z + 1,
                      hipBoundaryModeClamp);
    s_next[tz][ty + 2][tx + 0] = val;
    // Bottom-right
    surf2DLayeredread(&val, dog_octave, (x + 1) * sizeof(float), y + 1, z + 1,
                      hipBoundaryModeClamp);
    s_next[tz][ty + 2][tx + 2] = val;
    __syncthreads();

    // Make this check first.
    const auto on_edge =
        [&edge_ratio_thres](
            const volatile decltype(s_curr) s_data,  //
            auto tx, auto ty, auto tz) -> bool {
      //   const auto H = hessian(I, Point2i{x, y});
      //   return square(H.trace()) * edge_ratio >=
      //          square(edge_ratio + 1.f) * std::abs(H.determinant());
      return false;
    };

    if (on_edge(s_curr, tx, ty, tz))
    {
      ext_map[gi] = 0;
      return;
    }

    // Now the most expensive check.
    auto val_ext = val;
    if (val > 0)
    {
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = max(val_ext, s_prev[tz][ty + dy][tx + dx]);
        }
      }
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = max(val_ext, s_curr[tz][ty + dy][tx + dx]);
        }
      }
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = max(val_ext, s_next[tz][ty + dy][tx + dx]);
        }
      }
    }
    else
    {
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = min(val_ext, s_prev[tz][ty + dy][tx + dx]);
        }
      }
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = min(val_ext, s_curr[tz][ty + dy][tx + dx]);
        }
      }
#pragma unroll
      for (auto dy = 0; dy <= 2; ++dy)
      {
#pragma unroll
        for (auto dx = 0; dx <= 2; ++dx)
        {
          val_ext = min(val_ext, s_next[tz][ty + dy][tx + dx]);
        }
      }
    }

    auto extremum_type = std::int8_t{};
    if (val == val_ext && val > 0)
      extremum_type = 1;
    else if (val == val_ext && val < 0)
      extremum_type = -1;

    ext_map[gi] = extremum_type;
  }

  auto compute_dog_octave(const Octave<float>& gaussians, Octave<float>& dogs)
      -> void
  {
    if (dogs.width() != gaussians.width() ||
        dogs.height() != gaussians.height() ||
        dogs.scale_count() != gaussians.scale_count() - 1)
      throw std::runtime_error{"Invalid octave sizes!"};

    if (!gaussians.surface_object().initialized())
      throw std::runtime_error{"Gaussian surface object is uninitialized!"};

    if (!dogs.surface_object().initialized())
      dogs.init_surface();

    static constexpr auto threadsperBlock = dim3(32, 32, 1);
    static const auto numBlocks =
        dim3((dogs.width() + threadsperBlock.x - 1) / threadsperBlock.x,
             (dogs.height() + threadsperBlock.y - 1) / threadsperBlock.y,
             (dogs.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
    dog<<<numBlocks, threadsperBlock>>>(gaussians.surface_object(),
                                        dogs.surface_object(), dogs.width(),
                                        dogs.height(), dogs.scale_count());
  }

  auto compute_scale_space_extremum_map(
      const Octave<float>& dogs,
      MultiArrayView<std::int8_t, 3, RowMajorStrides>& extremum_map,
      float min_extremum_abs_value, float edge_ratio_thres) -> void
  {
    if (extremum_map.width() != dogs.width() ||
        extremum_map.height() != dogs.height() ||
        extremum_map.depth() != dogs.scale_count())
      throw std::runtime_error{"Invalid octave sizes!"};

    if (!dogs.surface_object().initialized())
      throw std::runtime_error{"DoG surface object is uninitialized!"};

    static constexpr auto threadsperBlock = dim3(tile_x, tile_y, tile_z);
    static const auto numBlocks =
        dim3((dogs.width() + threadsperBlock.x - 1) / threadsperBlock.x,
             (dogs.height() + threadsperBlock.y - 1) / threadsperBlock.y,
             (dogs.scale_count() + threadsperBlock.z - 1) / threadsperBlock.z);
    local_scale_space_extremum<<<numBlocks, threadsperBlock>>>(
        dogs.surface_object(), extremum_map.data(),       //
        dogs.width(), dogs.height(), dogs.scale_count(),  //
        extremum_map.padded_width(), min_extremum_abs_value, edge_ratio_thres);
  }

}  // namespace DO::Shakti::Cuda
